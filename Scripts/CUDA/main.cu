#include "hip/hip_runtime.h"
#include "aligner.h"
#include <hip/hip_runtime_api.h>
#include <time.h>
//__global__ void kernel(float* MAT,char* a,char* b,int *k){
__global__ void kernel(float* MAT,char** a,int *k,double *len){
  int col = blockIdx.y*blockDim.y+threadIdx.y;
  int row = blockIdx.x*blockDim.x+threadIdx.x;
  int l = int(*len);
  if(col < l && row < l){
    if(col>row){
      MAT[row*l+col] = aligner::kmdist(a[row],a[col],k);
    }
  }
  __syncthreads();
}
//aligner::compare(ex1,ex2,m);
void compare(float* MAT,float* HMAT,double size,double len,char* r[],int k);
unsigned long vecsize(double f);
int main(int argc, char const *argv[]) {
  string file = argv[1];
  int K = atoi(argv[2]);
  aligner objAl(file);
  double len = objAl.length();
  objAl.getReads();
  double size = len*len;
  //vector<float> h_mat(size);
  float* h_mat;
  h_mat = (float*)malloc(int(size*sizeof(float)));
  memset(h_mat,0,int(size*sizeof(float)));
  float* d_mat;
  checkCudaErrors(hipMalloc((void**)&d_mat,int(size*sizeof(float))));
  checkCudaErrors(hipMemcpy(d_mat,h_mat,int(size*sizeof(float)),hipMemcpyHostToDevice));
  compare(d_mat,h_mat,size,len,objAl.h_reads,K);
  for(int i = 0;i<int(len);i++){
    for(int j = 0;j<(len);j++){
      printf("%f  ", h_mat[i*int(len)+j]);
    }
    printf("\n");
  }
  return 0;
}
unsigned long vecsize(double f){
  unsigned long s = 0;
  for(int i = 0;i<f;i++){s+=i;}
  return s;
}
//Call to the global function and make everything
void compare(float* MAT,float* HMAT,double size,double len,char* r[],int k){
  char **d_reads, **d_tmp;
  checkCudaErrors(hipMalloc((void**)&d_reads,len*sizeof(char*)));
  d_tmp = (char**)malloc(len*sizeof(char*));
  int slen = 0;
  for(int i=0;i<len;i++){
    slen = strlen(r[i]);
    checkCudaErrors(hipMalloc(&(d_tmp[i]),slen*sizeof(char)));
    checkCudaErrors(hipMemcpy(d_tmp[i],r[i],slen*sizeof(char),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_reads+i,&(d_tmp[i]),sizeof(char*),hipMemcpyHostToDevice));
  }
  int *d_k;
  int* ptr_max_len = &k;
  checkCudaErrors(hipMalloc((void**)&d_k,int(sizeof(int))));
  checkCudaErrors(hipMemcpy(d_k,ptr_max_len,int(sizeof(int)),hipMemcpyHostToDevice));
  double *d_len;
  double* d_tmp_len = &len;
  checkCudaErrors(hipMalloc((void**)&d_len,int(sizeof(double))));
  checkCudaErrors(hipMemcpy(d_len,d_tmp_len,int(sizeof(double)),hipMemcpyHostToDevice));
  dim3 threadsPerBlock(len, len);
  dim3 blocksPerGrid(1, 1);
  if (len*len > 256){
    threadsPerBlock.x = 16;
    threadsPerBlock.y = 16;
    blocksPerGrid.x = ceil(double(len)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(len)/double(threadsPerBlock.y));
  }
  //para tomar el tiempo
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //funcion paralela
  hipEventRecord(start,0);
  kernel<<<blocksPerGrid,threadsPerBlock>>>(MAT,d_reads,d_k,d_len);
  hipDeviceSynchronize();
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float timer = 0;
  hipEventElapsedTime(&timer,start,stop);
  cout << "Elapsed parallel time:" << timer/1000 << "seconds" << endl;
  hipDeviceSynchronize();
  checkCudaErrors(hipMemcpy(HMAT,MAT,int(size*sizeof(float)),hipMemcpyDeviceToHost));
  hipDeviceSynchronize();
}
