#include "hip/hip_runtime.h"
#include "aligner.h"
#include <hip/hip_runtime_api.h>
#include <time.h>
//__global__ void kernel(float* MAT,char* a,char* b,int *k){
__global__ void kernel(float* MAT,char** a,int *k,double *len){
  int col = blockIdx.y*blockDim.y+threadIdx.y;
  int row = blockIdx.x*blockDim.x+threadIdx.x;
  if(col>row){
    MAT[row*int(*len)+col] = aligner::kmdist(a[row],a[col],k);
  }
}
//aligner::compare(ex1,ex2,m);
void compare(float* MAT,float* HMAT,double size,double len,char* r[],int k);
unsigned long vecsize(double f);
int main(int argc, char const *argv[]) {
  string file = argv[1];
  int K = atoi(argv[2]);
  aligner objAl(file);
  double len = objAl.length();
  objAl.getReads();
  double size = len*len;
  //vector<float> h_mat(size);
  float* h_mat;
  h_mat = (float*)malloc(int(size*sizeof(float)));
  //dev_array<float> d_mat(size);
  float* d_mat;
  checkCudaErrors(hipMalloc((void**)&d_mat,int(size*sizeof(float))));
  checkCudaErrors(hipMemcpy(d_mat,h_mat,int(size*sizeof(float)),hipMemcpyHostToDevice));
  compare(d_mat,h_mat,size,len,objAl.h_reads,K);
  for(int i = 0;i<int(len);i++){
    for(int j = 0;j<(len);j++){
      printf("%f  ", h_mat[i*int(len)+j]);
    }
    printf("\n");
  }
  return 0;
}
unsigned long vecsize(double f){
  unsigned long s = 0;
  for(int i = 0;i<f;i++){s+=i;}
  return s;
}
//Call to the global function and make everything
void compare(float* MAT,float* HMAT,double size,double len,char* r[],int k){
  char **d_reads, **d_tmp;
  checkCudaErrors(hipMalloc((void**)&d_reads,len*sizeof(char*)));
  d_tmp = (char**)malloc(len*sizeof(char*));
  int slen = 0;
  for(int i=0;i<len;i++){
    slen = strlen(r[i]);
    checkCudaErrors(hipMalloc(&(d_tmp[i]),slen*sizeof(char)));
    checkCudaErrors(hipMemcpy(d_tmp[i],r[i],slen*sizeof(char),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_reads+i,&(d_tmp[i]),sizeof(char*),hipMemcpyHostToDevice));
  }
  int *d_k;
  int* ptr_max_len = &k;
  checkCudaErrors(hipMalloc((void**)&d_k,int(sizeof(int))));
  checkCudaErrors(hipMemcpy(d_k,ptr_max_len,int(sizeof(int)),hipMemcpyHostToDevice));
  double *d_len;
  double* d_tmp_len = &len;
  checkCudaErrors(hipMalloc((void**)&d_len,int(sizeof(double))));
  checkCudaErrors(hipMemcpy(d_len,d_tmp_len,int(sizeof(double)),hipMemcpyHostToDevice));
  dim3 threadsPerBlock(len, len);
  dim3 blocksPerGrid(1, 1);
  if (len*len > 1024){
    threadsPerBlock.x = 32;
    threadsPerBlock.y = 32;
    blocksPerGrid.x = ceil(double(len)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(len)/double(threadsPerBlock.y));
  }
  //para tomar el tiempo
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //funcion paralela
  hipEventRecord(start,0);
  kernel<<<blocksPerGrid,threadsPerBlock>>>(MAT,d_reads,d_k,d_len);
  hipDeviceSynchronize();
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float timer = 0;
  hipEventElapsedTime(&timer,start,stop);
  cout << "Elapsed parallel time:" << timer/1000 << "seconds" << endl;
  checkCudaErrors(hipMemcpy(HMAT,MAT,int(size*sizeof(float)),hipMemcpyDeviceToHost));
  hipDeviceSynchronize();
}
//MAT.get(&HMAT[0],size);
/*
int max_len, num_str;
num_str = 6;
char* tmp[num_str];
max_len = k;
int *d_max_len;
int* ptr_max_len = &k;
checkCudaErrors(hipMalloc((void**)&d_max_len,sizeof(int)));
checkCudaErrors(hipMemcpy(d_max_len,ptr_max_len,sizeof(int),hipMemcpyHostToDevice));
for(int i=0;i<num_str;i++){
  tmp[i] = (char*) malloc(max_len*sizeof(char));
}
char* ex1 = "abb";
char* ex2 = "abd";
char* ex3 = "abc";
char* ex4 = "aaa";
char* ex5 = "aab";
char* ex6 = "bbb";
char* ex7 = "bba";
tmp[0] = ex1;
tmp[1] = ex2;
tmp[2] = ex3;
tmp[3] = ex4;
tmp[4] = ex5;
tmp[5] = ex6;
char* ex, *d_ex1, *d_ex2;
ex = (char*) malloc(max_len*num_str*sizeof(char));
ex1 = (char*) malloc(max_len*num_str*sizeof(char));
int n = 0;
for(int i=0;i<num_str;i++){
  for(int j = 0;j<max_len;j++){
    ex[n] = tmp[i][j];
    ex1[n] = tmp[i][j];
    n++;
  }
}
int s = max_len*num_str*sizeof(char);
checkCudaErrors(hipMalloc((void**)&d_ex1,s));
checkCudaErrors(hipMemcpy(d_ex1,ex,s,hipMemcpyHostToDevice));
checkCudaErrors(hipMalloc((void**)&d_ex2,s));
checkCudaErrors(hipMemcpy(d_ex2,ex1,s,hipMemcpyHostToDevice));
double *d, *d_d;
d = (double*)malloc(int(sizeof(double)));
*d = 1.0;
checkCudaErrors(hipMalloc((void**)&d_d,int(sizeof(double))));
checkCudaErrors(hipMemcpy(d_d,d,int(sizeof(double)),hipMemcpyHostToDevice));
//lo anterior funciona
char** d_tmp;
checkCudaErrors(hipMalloc((void**)&d_tmp,s));
checkCudaErrors(hipMemcpy(d_tmp,tmp,s,hipMemcpyHostToDevice));
*/
