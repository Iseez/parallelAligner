#include "hip/hip_runtime.h"
#include "aligner.h"
#include <hip/hip_runtime_api.h>
#include <time.h>
//__global__ void kernel(float* MAT,char* a,char* b,int *k){
__global__ void kernel(float* MAT,char** a,int *k){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  float ff = aligner::kmdist(a[0],a[0],k);
  printf("%f\n", ff);
  MAT[index] = index;
}
//aligner::compare(ex1,ex2,m);
void compare(dev_array<float> MAT,vector<float> HMAT,long size,double len,char* r[],int k);
unsigned long vecsize(double f);
int main(int argc, char const *argv[]) {
  string file = argv[1];
  int K = atoi(argv[2]);
  aligner objAl(file);
  double len = objAl.length();
  objAl.getReads();
  unsigned long size = vecsize(len);
  vector<float> h_mat(size);
  dev_array<float> d_mat(size);
  compare(d_mat,h_mat,size,len,objAl.h_reads,K);
  return 0;
}
unsigned long vecsize(double f){
  unsigned long s = 0;
  for(int i = 0;i<f;i++){s+=i;}
  return s;
}
//Call to the global function and make everything
void compare(dev_array<float> MAT,vector<float> HMAT,long size,double len,char* r[],int k){
  char **d_reads, **d_tmp;
  checkCudaErrors(hipMalloc((void**)&d_reads,len*sizeof(char*)));
  d_tmp = (char**)malloc(len*sizeof(char*));
  int slen = 0;
  for(int i=0;i<len;i++){
    slen = strlen(r[i]);
    checkCudaErrors(hipMalloc(&(d_tmp[i]),slen*sizeof(char)));
    checkCudaErrors(hipMemcpy(d_tmp[i],r[i],slen*sizeof(char),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_reads+i,&(d_tmp[i]),sizeof(char*),hipMemcpyHostToDevice));
  }
  int *d_k;
  int* ptr_max_len = &k;
  checkCudaErrors(hipMalloc((void**)&d_k,int(sizeof(int))));
  checkCudaErrors(hipMemcpy(d_k,ptr_max_len,int(sizeof(int)),hipMemcpyHostToDevice));
  int threads = size;
  int blocks = 1;
  if(size > 1024){
    blocks = ceil(size/1024);
  }
  //para tomar el tiempo
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //funcion paralela
  hipEventRecord(start,0);
  //kernel<<<1,1>>>(MAT.getData(),d_ex1,d_ex2,d_max_len);
  kernel<<<1,1>>>(MAT.getData(),d_reads,d_k);
  hipDeviceSynchronize();
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float timer = 0;
  hipEventElapsedTime(&timer,start,stop);
  cout << "Elapsed parallel time:" << timer/1000 << "seconds" << endl;
}
//MAT.get(&HMAT[0],size);
/*
int max_len, num_str;
num_str = 6;
char* tmp[num_str];
max_len = k;
int *d_max_len;
int* ptr_max_len = &k;
checkCudaErrors(hipMalloc((void**)&d_max_len,sizeof(int)));
checkCudaErrors(hipMemcpy(d_max_len,ptr_max_len,sizeof(int),hipMemcpyHostToDevice));
for(int i=0;i<num_str;i++){
  tmp[i] = (char*) malloc(max_len*sizeof(char));
}
char* ex1 = "abb";
char* ex2 = "abd";
char* ex3 = "abc";
char* ex4 = "aaa";
char* ex5 = "aab";
char* ex6 = "bbb";
char* ex7 = "bba";
tmp[0] = ex1;
tmp[1] = ex2;
tmp[2] = ex3;
tmp[3] = ex4;
tmp[4] = ex5;
tmp[5] = ex6;
char* ex, *d_ex1, *d_ex2;
ex = (char*) malloc(max_len*num_str*sizeof(char));
ex1 = (char*) malloc(max_len*num_str*sizeof(char));
int n = 0;
for(int i=0;i<num_str;i++){
  for(int j = 0;j<max_len;j++){
    ex[n] = tmp[i][j];
    ex1[n] = tmp[i][j];
    n++;
  }
}
int s = max_len*num_str*sizeof(char);
checkCudaErrors(hipMalloc((void**)&d_ex1,s));
checkCudaErrors(hipMemcpy(d_ex1,ex,s,hipMemcpyHostToDevice));
checkCudaErrors(hipMalloc((void**)&d_ex2,s));
checkCudaErrors(hipMemcpy(d_ex2,ex1,s,hipMemcpyHostToDevice));
double *d, *d_d;
d = (double*)malloc(int(sizeof(double)));
*d = 1.0;
checkCudaErrors(hipMalloc((void**)&d_d,int(sizeof(double))));
checkCudaErrors(hipMemcpy(d_d,d,int(sizeof(double)),hipMemcpyHostToDevice));
//lo anterior funciona
char** d_tmp;
checkCudaErrors(hipMalloc((void**)&d_tmp,s));
checkCudaErrors(hipMemcpy(d_tmp,tmp,s,hipMemcpyHostToDevice));
*/
